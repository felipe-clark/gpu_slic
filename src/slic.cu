#include <stdio.h>
#include <opencv2/opencv.hpp>
#include "../include/slic.h"



int main(int argc, char** argv)
{
    if(argc !=2)
    {
        printf("Invalid number of parameters (2 expected)\n");
        return -1;
    }

    char* imageName = argv[1];

    cv::Mat image;
    image = cv::imread(imageName, cv::IMREAD_COLOR);

    if(!image.data)
    {
        printf("Could not open image\n");
        return -2;
    }

    cv::Size size(width, height);
    cv::Mat resized;
    cv::resize(image, resized, size);

    if(!resized.isContinuous())
    {
        printf("OpenCV is being difficult. Sorry :,(. Suiciding.\n");
        return -3;
    }

    cv::imwrite("./resized_image.tif", resized);

    cv::Mat lab_image;
    cv::cvtColor(resized, lab_image, cv::COLOR_BGR2Lab);

    printf("Bytes: %lu\n", lab_image.total()*lab_image.channels());

    // Preparations for Kernel invokation
    unsigned char* d_output;
    unsigned char* d_image;
    spixel_data* d_spixel_data;
    ownership_data* d_ownership_data;

    int img_byte_size = 3 * width * height * sizeof(unsigned char);
    int spix_byte_size = spixel_size * spixel_size * sizeof(spixel_data);
    int own_byte_size = width * height * sizeof(ownership_data);

    hipMalloc(&d_output, img_byte_size);
    hipMalloc(&d_image, img_byte_size);
    hipMalloc(&d_spixel_data, spix_byte_size);
    hipMalloc(&d_ownership_data, own_byte_size);

    cv::Mat h_result = lab_image.clone();
    hipMemcpy(d_image, lab_image.data, img_byte_size, hipMemcpyHostToDevice);

    spixel_data* h_spixel_data = (spixel_data*)malloc(spix_byte_size);
    initialize_centers(h_spixel_data);
    hipMemcpy(d_spixel_data, h_spixel_data, spix_byte_size, hipMemcpyHostToDevice);

    ownership_data* h_ownership_data = (ownership_data*)malloc(own_byte_size);
    initialize_ownership(h_ownership_data);
    hipMemcpy(d_ownership_data, h_ownership_data, own_byte_size, hipMemcpyHostToDevice);




    // -------------------- The Kernel magic --------------------

    // Configure how to launch the Matrix Add CUDA Kernel
    dim3 threadsPerBlock( 32, 32 ) ;

    // Block split as recommended in the assignment sheet
    int blockPerGridX = (width + threadsPerBlock.x-1)/threadsPerBlock.x;
    int blockPerGridY = (height + threadsPerBlock.y-1)/threadsPerBlock.y;
    int blockPerGridZ = 1;

    dim3 blocksPerGrid( blockPerGridX, blockPerGridY,  blockPerGridZ );

    //kernelOverPixels<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_output);
    cummulativeCount<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_ownership_data, d_spixel_data);
    hipDeviceSynchronize();


    // --- START KERNEL 2 ---

    dim3 spx_threadsPerBlock(32, 32);

    int spx_blockPerGridX = (spixel_width + spx_threadsPerBlock.x-1)/spx_threadsPerBlock.x;
    int spx_blockPerGridY = (spixel_height + spx_threadsPerBlock.y-1)/spx_threadsPerBlock.y;
    int spx_blockPerGridZ = 1;

    dim3 spx_blocksPerGrid( spx_blockPerGridX, spx_blockPerGridY, spx_blockPerGridZ);
    averaging<<<spx_blocksPerGrid, spx_threadsPerBlock>>>(d_spixel_data);



    hipMemcpy(h_result.data, d_image, img_byte_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_spixel_data, d_spixel_data, spix_byte_size, hipMemcpyDeviceToHost);

    

    hipDeviceReset();

    cv::Mat rgb_result_image;

    test_mark_spixel_centers(h_result.data, h_spixel_data);
    test_block_spixels(h_result.data, h_ownership_data, h_spixel_data);

    cv::cvtColor(h_result, rgb_result_image, cv::COLOR_Lab2BGR);

    cv::imwrite("./processed_image.jpg", rgb_result_image);

    printf("SUCCESS!\n");

}


void initialize_centers(spixel_data* spx_data)
{
    for (int i = 0; i < spixel_width; i++)
    {
        for(int j = 0; j < spixel_height; j++)
        {
            int index = j * spixel_width + i;
            spx_data[index].l = 127;
            spx_data[index].a = 127;
            spx_data[index].b = 127;
            spx_data[index].x = (2 * spixel_size * i + spixel_size)/2;
            spx_data[index].y = (2 * spixel_size * j + spixel_size)/2;

            spx_data[index].l_acc = 0;
            spx_data[index].a_acc = 0;
            spx_data[index].b_acc = 0;
            spx_data[index].n_pix = 0;
        }
    }
}

void initialize_ownership(ownership_data* h_ownership_data)
{
    for (int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int own_index = y * width + x;


            int i = x/spixel_size;
            int j = y/spixel_size;

            h_ownership_data[own_index].i = i;
            h_ownership_data[own_index].j = j;
        }
    }
}

void test_mark_spixel_centers(unsigned char* h_image, const spixel_data* spx_data)
{
    for (int i = 0; i < spixel_width; i++)
    {
        for(int j = 0; j < spixel_height; j++)
        {
            int index = j * spixel_width + i;
            int x = spx_data[index].x;
            int y = spx_data[index].y;

            int img_index = 3 * (y * width + x);

            h_image[img_index] = 0;
        }
    }
}

void test_block_spixels(unsigned char* h_image, ownership_data* h_ownership_data, spixel_data* h_spixel_data)
{
    for (int x = 0; x < width; x++)
    {
        for(int y = 0; y < height; y++)
        {
            int own_index = y * width + x;
            int img_index = 3 * own_index;

            // Funky colors ;)
            //h_image[img_index+1] = h_ownership_data[own_index].i*10;
            //h_image[img_index+2] = h_ownership_data[own_index].j*10;

            int spx_index = h_ownership_data[own_index].j * spixel_width + h_ownership_data[own_index].i;
            h_image[img_index + 0] = h_spixel_data[spx_index].l;
            h_image[img_index + 1] = h_spixel_data[spx_index].a;
            h_image[img_index + 2] = h_spixel_data[spx_index].b;
        }
    }
}
