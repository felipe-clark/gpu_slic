#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include "../include/slic.h"

__device__ __constant__ float slic_factor;

void initializeSlicFactor()
{
    const float * slic_factor_hp = &slic_factor_h;
    hipError_t cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(slic_factor), slic_factor_hp, sizeof(float));
}

__global__ void k_cumulativeCountOrig(const pix_data* d_pix_data, const own_data* d_own_data, spx_data* d_spx_data)
{
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {
	    printf("k\n");
    }

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < pix_height && x < pix_width) 
    {
        int pix_index = y * pix_width + x;
        int i = d_own_data[pix_index].i;
        int j = d_own_data[pix_index].j;
        int spx_index = j * spx_width + i;

        atomicAdd(&(d_spx_data[spx_index].l_acc), d_pix_data[pix_index].l);
        atomicAdd(&(d_spx_data[spx_index].a_acc), d_pix_data[pix_index].a);
        atomicAdd(&(d_spx_data[spx_index].b_acc), d_pix_data[pix_index].b);
        atomicAdd(&(d_spx_data[spx_index].num), 1);
    }
}

__global__ void k_cumulativeCountOpt1(const pix_data* d_pix_data, const own_data* d_own_data, spx_data* d_spx_data)
{
    //if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    //{
	    //printf("K\n");
    //}

    // If we do 16 instead of 8, only have enough memory for a short, not an int,
    // and 16*32*255 does not fit in a short
    __shared__ int acc[4][3][3][8][32]; //LAB+count, 3x3 neighbors, 8x32 values

    int tidx=threadIdx.x;
    int tidy=threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    for (int nx=0;nx<3;++nx) for (int ny=0;ny<3;++ny) for(int c=0;c<4;++c) acc[c][ny][nx][tidy][tidx]=0;

    int i_center = blockIdx.x * blockDim.x / spx_size;
    int j_center = blockIdx.y * blockDim.y / spx_size;
    int pix_index = y * pix_width + x;
    int i = d_own_data[pix_index].i;
    int j = d_own_data[pix_index].j;
    int nx = (i<i_center) ? 0 : ((i>i_center) ? 2 : 1);
    int ny = (j<j_center) ? 0 : ((j>j_center) ? 2 : 1);
    acc[0][ny][nx][tidy][tidx] = d_pix_data[pix_index].l;
    acc[1][ny][nx][tidy][tidx] = d_pix_data[pix_index].a;
    acc[2][ny][nx][tidy][tidx] = d_pix_data[pix_index].b;
    acc[3][ny][nx][tidy][tidx] = 1;
   
    __syncthreads();

    // Collapse over X
    for (int step=1; step<32; step *= 2)
    {
        if (tidx % (2*step) == 0)
        {
            for (int ny=0; ny<3; ny++)
            for (int nx=0; nx<3; nx++)
            for (int c=0; c<4; c++)
            acc[c][ny][nx][tidy][tidx] += acc[c][ny][nx][tidy][tidx + step];
        }
    }

    // Is this ok? See https://stackoverflow.com/questions/6666382/can-i-use-syncthreads-after-having-dropped-threads
    // TODO: Use these threads for nx, ny, c loop
    if (tidy != 0) return;
    __syncthreads();

    if (tidx>=8) return;
    // Collapse over Y
    for (int step=1; step<8; step *= 2)
    {
        if (tidx % (2*step) == 0)
        {
            for (int ny=0; ny<3; ny++)
            for (int nx=0; nx<3; nx++)
            for (int c=0; c<4; c++)
            acc[c][ny][nx][tidx][0] += acc[c][ny][nx][tidx + step][0];
        }
    }

    // Now, acc[c][ny][nx][0][0] has the values we need
    // but where do we write them to?
    
    // Just one warp so no syncThreads (TODO)
    if (tidx != 0) return;

    for (int ny=0; ny<3; ny++)
    {
        int j = j_center + ny - 1;
	if (j<0 || j>=spx_height) continue;
        for (int nx=0; nx<3; nx++)
        {
            int i = i_center + nx - 1;
            if (i<0 || i>=spx_width) continue;

            int spx_index = j * spx_width + i;


	    //if (blockIdx.x ==0 && blockIdx.y == 0)
	    //printf("A:%d %d %d %u %u %u %u\n", i_center, j_center, spx_index, acc[0][ny][nx][0][0], acc[1][ny][nx][0][0], acc[2][ny][nx][0][0], acc[3][ny][nx][0][0]); 
            
	    atomicAdd(&(d_spx_data[spx_index].l_acc), (int)acc[0][ny][nx][0][0]);
            atomicAdd(&(d_spx_data[spx_index].a_acc), (int)acc[1][ny][nx][0][0]);
            atomicAdd(&(d_spx_data[spx_index].b_acc), (int)acc[2][ny][nx][0][0]);
            atomicAdd(&(d_spx_data[spx_index].num),   (int)acc[3][ny][nx][0][0]);
	    
	    //if (blockIdx.x==0 && blockIdx.y==0)
	    //{
	       //printf("C:%u %u %u %u\n", d_spx_data[spx_index].l_acc, d_spx_data[spx_index].a_acc, d_spx_data[spx_index].b_acc, d_spx_data[spx_index].num); 
	       //printf("J\n");
	    //}
        }
    }
}

__global__ void k_averaging(spx_data* d_spx_data)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < spx_width && j < spx_height)
    {
        int spx_index = j * spx_width + i;
        d_spx_data[spx_index].l = d_spx_data[spx_index].l_acc / d_spx_data[spx_index].num;
        d_spx_data[spx_index].a = d_spx_data[spx_index].a_acc / d_spx_data[spx_index].num;
        d_spx_data[spx_index].b = d_spx_data[spx_index].b_acc / d_spx_data[spx_index].num;
    }
}

__global__ void k_ownership(const pix_data* d_pix_data, own_data* d_own_data, const spx_data* d_spx_data)
{
    float min_dist = 10E99;// max_float;
    int min_i = 0;
    int min_j = 0;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < pix_height && x < pix_width) 
    {
        int pix_index = y * pix_width + x;
        int i_center = x/spx_size;
        int j_center = y/spx_size;

        int l = d_pix_data[pix_index].l;
        int a = d_pix_data[pix_index].a;
        int b = d_pix_data[pix_index].b;

        for (int i = i_center - window_size; i <= i_center + window_size; i++)
        {
            if (i < 0 || i >= spx_width) continue;

            for(int j = j_center - window_size; j <= j_center + window_size; j++)
            {
                if (j < 0 || j >= spx_height) continue;

                int spx_index = j * spx_width + i;
                int l_dist = l-(int)(d_spx_data[spx_index].l);
                l_dist *= l_dist;
                int a_dist = a-(int)(d_spx_data[spx_index].a);
                a_dist *= a_dist;
                int b_dist = b-(int)(d_spx_data[spx_index].b);
                b_dist *= b_dist;
                int dlab = l_dist + a_dist + b_dist;

                int x_dist = x-(int)d_spx_data[spx_index].x;
                x_dist *= x_dist;
                int y_dist = y-(int)d_spx_data[spx_index].y;
                y_dist *= y_dist;
                int dxy = x_dist + y_dist;

                float D = dlab + slic_factor * dxy;

                if (D < min_dist)
                {
                    min_dist = D;
                    min_i = i;
                    min_j = j;
                }
            }
        }

        d_own_data[pix_index].i = min_i;
        d_own_data[pix_index].j = min_j;

        //d_own_data[pix_index].i = (i_center / 4) * 4;
        //d_own_data[pix_index].j = (j_center / 4) * 4;
    }
}

__global__ void k_reset(spx_data* d_spx_data)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < spx_width && j < spx_height)
    {
        int spx_index = j * spx_width + i;
        d_spx_data[spx_index].l_acc = 0;
        d_spx_data[spx_index].a_acc = 0;
        d_spx_data[spx_index].num = 0;
    }
}
